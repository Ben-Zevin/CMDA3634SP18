#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int d_modprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int d_modExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = d_modprod(aExpb, z, p);
    z = d_modprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void kernelFindKeyPoint(unsigned int *d, unsigned int p, unsigned int g, unsigned int h)
{
	int threadId = threadIdx.x;
	int blockId = blockIdx.x;
	int Nblock = blockDim.x;

	int id = threadId +blockId*Nblock;

	if(d_modExp(g, id, p)==h)
	{
			d[0] = id;
	}
}


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  int Nthreads = atoi(argv[1]);

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE *key, *message;
 // key = fopen("public_key.txt", "r");
  key = fopen("bonus_public_key.txt", "r");
  fscanf(key, "%u" "%u" "%u" "%u", &n, &p, &g, &h);
  fclose(key);
 

//  message = fopen("message.txt", "r");
  message = fopen("bonus_message.txt", "r");
  fscanf(message, "%d", &Nints);
  unsigned int *Zmessage = 
      (unsigned int *) malloc(Nints*sizeof(unsigned int)); 
  
  //storage for extra encryption coefficient 
  unsigned int *a = 
      (unsigned int *) malloc(Nints*sizeof(unsigned int)); 

  for (int i=0; i < Nints; i++)
  {
		  fscanf(message, "%u" "%u", &Zmessage[i], &a[i]);
  }
  fclose(message);



  // find the secret key
 // if (x==0 || modExp(g,x,p)!=h) {
 //   printf("Finding the secret key...\n");
    double startTime = clock();
 //   for (unsigned int i=0; i<p-1; i++) {
 //     if (modExp(g,i+1,p)==h) {
 //       printf("Secret key found! x = %u \n", i+1);
 //       x=i+1;
 //     } 
 //   }
	unsigned int *h_key = (unsigned int*) malloc(1*sizeof(unsigned int));

    unsigned int *d_key;
	hipMalloc(&d_key,1*sizeof(unsigned int));

	dim3 B(Nthreads, 1, 1);
	dim3 G((p+Nthreads-1)/Nthreads,1,1);

	kernelFindKeyPoint <<<G ,B>>>(d_key, p, g, h);

	hipDeviceSynchronize();

    double endTime = clock();

	hipMemcpy(h_key,d_key,1*sizeof(unsigned int),hipMemcpyDeviceToHost);
	x = h_key[0];

	ElGamalDecrypt(Zmessage,a,Nints,p,x);
	
	unsigned int charsPerInt = (n-1)/8;
	unsigned int Nchars = Nints*charsPerInt;
	
	int bufferSize = 1024;
    unsigned char *newString = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
	

   
	convertZToString(Zmessage, Nints, newString, Nchars);
	
    printf("Decrypted Message = \"%s\"\n", newString);
    printf("\n");


    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);

	hipFree(d_key);

	free(h_key);
	free(Zmessage);
	free(a);
	free(newString);
  return 0;
}
