#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void kernelHelloWorld() {

	int thread = threadIdx.x;
	int block = blockIdx.x;

	printf("Hello World from thread %d of block %d!\n", thread, block);
}

int main(int argc, char** argv){

	int Nblocks = 10;
	int Nthreads = 3;

	kernelHelloWorld <<< Nblocks, Nthreads >>> ();

	hipDeviceSynchronize();

	return 0;

}

