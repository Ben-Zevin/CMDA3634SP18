#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

// This is my DEVICE function
// __global__ means this function is visible to the host
__global__  void kernelHelloWorld() {

		printf("HEllo World!\n");

}

int main(int argc, char** argv) {


		int Nblocks = 10; //number of blocks
		int Nthreads = 3; //number of threads
		
		// run the function 'kernalHellowWorld' on the DEVICE
		kernelHelloWorld <<< Nblocks, Nthreads >>> ();



}
